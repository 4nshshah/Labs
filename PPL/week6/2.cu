#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>

__global__ void occur(char *a, char *p, int *count, int n)
{
    int id = threadIdx.x;
    int c = 0;
    for (int i = id; i < id + n; i++)
        if (a[i] != p[i - id])
            c += 1;
    if (c == 0)
        atomicAdd(count, 1);
}

int main()
{
    char hA[100], *dA, hB[100], *dB;
    int c = 0;
    int *count = &c, result, *dC;
    printf("Enter the string and pattern:\n");
    scanf("%s", hA);
    printf("Enter the string and pattern:\n");
    scanf("%s", hB);

    hipMalloc((void **)&dA, strlen(hA) * sizeof(char));
    hipMalloc((void **)&dB, strlen(hB) * sizeof(char));
    hipMalloc((void **)&dC, sizeof(int));

    hipMemcpy(dA, hA, strlen(hA) * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, strlen(hB) * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(dC, count, sizeof(int), hipMemcpyHostToDevice);
    int n = strlen(hA) / strlen(hB);
    occur<<<1, n>>>(dA, dB, dC, n);
    hipMemcpy(&result, dC, sizeof(int), hipMemcpyDeviceToHost);
    printf("Count:%d\n", result);
}