#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define N 105

__global__ void vecAdd(int *a, int *b, int *c)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < N)
    {
        c[i] = a[i] + b[i];
    }
}

int main()
{
    int *a = (int *)malloc(N * sizeof(int));
    int *b = (int *)malloc(N * sizeof(int));
    int *c = (int *)malloc(N * sizeof(int));
    for (int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i;
    }

    int *da, *db, *dc;
    hipMalloc(&da, sizeof(a));
    hipMalloc(&db, sizeof(b));
    hipMalloc(&dc, sizeof(c));

    hipMemcpy(da, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(b), hipMemcpyHostToDevice);
    hipMemcpy(dc, c, sizeof(c), hipMemcpyHostToDevice);

    vecAdd<<<N, 1>>>(da, db, dc);

    hipMemcpy(c, dc, sizeof(c), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++)
        printf("%d\n", c[i]);

    // free(a);
    // free(b);
    // free(c);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
}
