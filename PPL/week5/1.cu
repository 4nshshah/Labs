#include "hip/hip_runtime.h"

#include <stdio.h>
__global__ void add(int *a, int *b, int *c)
{
    int i = threadIdx.x + (blockIdx.x * blockDim.x);
    c[i] = a[i] + b[i];
}

int main(void)
{
    int a[4] = {1, 2, 3, 4}, b[4] = {1, 2, 3, 4}, c[4];
    // host copies of variables a, b & c
    int *d_a = (int *)malloc(4 * sizeof(int));
    int *d_b = (int *)malloc(4 * sizeof(int));
    int *d_c = (int *)malloc(4 * sizeof(int));
    // device copies of variables a, b & c
    int size = sizeof(a) / sizeof(int);
    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    // Setup input values
    // Copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
    // Launch add() kernel on GPU
    add<<<4, 1>>>(d_a, d_b, d_c);
    // Copy result back to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < 4; i++)
        printf("Result: % d\n", c[i]);
    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}