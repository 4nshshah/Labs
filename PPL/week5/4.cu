#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Define the size of the vectors
#define N 10

// CUDA kernel to add two vectors
__global__ void vecAdd(float *a, float *b)
{
    // Get the global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < N)
    {
        b[id] = sinf(a[id]);
    }
}
// Main function
int main()
{
    // Allocate memory for the vectors on the host
    float *h_a = (float *)malloc(N * sizeof(float));
    float *h_b = (float *)malloc(N * sizeof(float));

    // Initialize the vectors
    for (int i = 0; i < N; i++)
    {
        h_a[i] = i;
    }

    // Allocate memory for the vectors on the device
    float *d_a, *d_b;
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));

    // Copy the vectors from the host to the device
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);

    // Launching the kernel:
    // N Blocks, 1 Thread
    vecAdd<<<N, 1>>>(d_a, d_b);
    // 1 block, N threads
    // vecAdd<<<1, N>>>(d_a, d_b);

    // Copy the result back from the device to the host
    hipMemcpy(h_b, d_b, N * sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < N; i++)
    {
        printf("%f\n", h_b[i]);
    }

    // Free the memory on the host
    free(h_a);
    free(h_b);

    // Free the memory on the device
    hipFree(d_a);
    hipFree(d_b);

    return 0;
}