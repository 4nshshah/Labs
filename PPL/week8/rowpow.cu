#include <stdio.h>
#include <hip/hip_runtime.h>
// Kernel function to replace each row of the matrix with the corresponding powers of its elements
__global__ void replace_rows_kernel(float *matrix, int m, int n)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m)
    {
        for (int j = 0; j < n; j++)
        {
            int index = row * n + j;
            matrix[index] = powf(matrix[index], row + 1);
        }
    }
}

int main()
{
    // Matrix dimensions
    int m = 3;
    int n = 3;

    // Matrix A
    float matrix[m][n];

    // Read matrix A from the user
    printf("Enter the elements of the matrix A:\n");
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            scanf("%f", &matrix[i][j]);
        }
    }

    // Allocate memory on device
    float *d_matrix;
    hipMalloc((void **)&d_matrix, m * n * sizeof(float));

    // Copy matrix A from host to device
    hipMemcpy(d_matrix, matrix, m * n * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int block_size = 256;
    int grid_size = (m + block_size - 1) / block_size;

    // Launch kernel
    replace_rows_kernel<<<grid_size, block_size>>>(d_matrix, m, n);

    // Copy matrix A from device to host
    hipMemcpy(matrix, d_matrix, m * n * sizeof(float), hipMemcpyDeviceToHost);

    // Print the modified matrix A
    printf("Modified matrix A:\n");
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            printf("%.2f ", matrix[i][j]);
        }
        printf("\n");
    }

    // Free memory on device
    hipFree(d_matrix);

    return 0;
}