
#include <hip/hip_runtime.h>
#include <stdio.h>

// Define matrix dimensions (M and N)
#define M 4
#define N 4

// Kernel to perform the 1's complement operation on non-border elements
__global__ void onesComplement(int *A, int *B, int rows, int cols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= 0 && row < rows && col >= 0 && col < cols)
    {
        // Check if the current element is a non-border element
        if (row > 0 && row < rows - 1 && col > 0 && col < cols - 1)
        {
            // Calculate the index for the current element
            int index = row * cols + col;
            // Calculate the 1's complement of the element and store it in B
            int number = A[index];
            int ulta = 0;
            for (int i = 0; number > 0; i++)
            {
                ulta *= 10;
                ulta += 1 - number % 2;
                number = number / 2;
            }
            B[index] = ulta;
        }
        else
        {
            // Copy border elements as-is
            B[row * cols + col] = A[row * cols + col];
        }
    }
}

int main()
{
    int A[M][N]; // Input matrix A
    int B[M][N]; // Output matrix B

    // Initialize matrix A with example values
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            A[i][j] = i * N + j;
        }
    }

    int *d_A, *d_B; // Device pointers for matrices A and B

    // Allocate memory on the GPU for matrices A and B
    hipMalloc((void **)&d_A, M * N * sizeof(int));
    hipMalloc((void **)&d_B, M * N * sizeof(int));

    // Copy matrix A from host to device
    hipMemcpy(d_A, A, M * N * sizeof(int), hipMemcpyHostToDevice);

    // Define thread and block dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch the CUDA kernel
    onesComplement<<<numBlocks, threadsPerBlock>>>(d_A, d_B, M, N);

    // Copy matrix B from device to host
    hipMemcpy(B, d_B, M * N * sizeof(int), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_A);
    hipFree(d_B);

    // Print matrix B (1's complement of non-border elements) in binary
    printf("Matrix B (1's complement of non-border elements in binary):\n");
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%d ", B[i][j]);
        }
        printf("\n");
    }

    return 0;
}
