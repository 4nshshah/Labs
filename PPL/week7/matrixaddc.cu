#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define SIZE 3

__global__ void matrixAddition(int *A, int *B, int *C, int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < width && col < width)
    {
        C[row * width + col] = A[row * width + col] + B[row * width + col];
    }
}

int main()
{
    int *h_A, *h_B, *h_C;
    int *d_A, *d_B, *d_C;

    // Allocate memory for matrices A, B, and C
    int matrixSize = SIZE * SIZE * sizeof(int);
    h_A = (int *)malloc(matrixSize);
    h_B = (int *)malloc(matrixSize);
    h_C = (int *)malloc(matrixSize);

    // Initialize matrices A and B
    for (int i = 0; i < SIZE * SIZE; i++)
    {
        h_A[i] = i;
        h_B[i] = i;
    }

    // Allocate memory in GPU
    hipMalloc((void **)&d_A, matrixSize);
    hipMalloc((void **)&d_B, matrixSize);
    hipMalloc((void **)&d_C, matrixSize);

    // Copy matrices A and B to GPU
    hipMemcpy(d_A, h_A, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, matrixSize, hipMemcpyHostToDevice);

    // Set up grid and block dimensions
    dim3 dimBlock(16, 16);
    dim3 dimGrid((SIZE + dimBlock.x - 1) / dimBlock.x, (SIZE + dimBlock.y - 1) / dimBlock.y);

    // Launch kernel
    matrixAddition<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, SIZE);

    // Copy result matrix C from GPU
    hipMemcpy(h_C, d_C, matrixSize, hipMemcpyDeviceToHost);

    // Print result matrix C
    for (int i = 0; i < SIZE; i++)
    {
        for (int j = 0; j < SIZE; j++)
        {
            printf("%d ", h_C[i * SIZE + j]);
        }
        printf("\n");
    }

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}