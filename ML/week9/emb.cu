#include "hip/hip_runtime.h"
#include <stdio.h>
#include <opencv2/opencv.hpp>

#define WIDTH 640
#define HEIGHT 480
#define KERNEL_SIZE 5

// CUDA kernel for 2D convolution
__global__ void convolution2D(int *input, int *mask, int *output, int width, int height, int kernelSize)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < height && col < width)
    {
        int sum = 0;
        int offset = kernelSize / 2;

        for (int i = 0; i < kernelSize; i++)
        {
            for (int j = 0; j < kernelSize; j++)
            {
                int r = row + i - offset;
                int c = col + j - offset;

                if (r >= 0 && r < height && c >= 0 && c < width)
                {
                    sum += input[r * width + c] * mask[i * kernelSize + j];
                }
            }
        }

        output[row * width + col] = sum;
    }
}

int main()
{
    cv::Mat inputImage = cv::imread("input.jpg", cv::IMREAD_GRAYSCALE); // Load the input image
    int *d_input, *d_mask, *d_output;
    int input[HEIGHT][WIDTH], output[HEIGHT][WIDTH];
    int mask[KERNEL_SIZE][KERNEL_SIZE] = {{1, 0, 0, 0, 0}, {1, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, -1, 0}, {0, 0, 0, 0, -1}}; // Embossing kernel

    if (inputImage.empty())
    {
        printf("Could not open or find the image.\n");
        return -1;
    }

    // Convert the OpenCV image to a grayscale input array
    for (int i = 0; i < HEIGHT; i++)
    {
        for (int j = 0; j < WIDTH; j++)
        {
            input[i][j] = static_cast<int>(inputImage.at<uchar>(i, j));
        }
    }

    // Allocate memory on the device
    hipMalloc((void **)&d_input, WIDTH * HEIGHT * sizeof(int));
    hipMalloc((void **)&d_mask, KERNEL_SIZE * KERNEL_SIZE * sizeof(int));
    hipMalloc((void **)&d_output, WIDTH * HEIGHT * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_input, input, WIDTH * HEIGHT * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, KERNEL_SIZE * KERNEL_SIZE * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 dimGrid((WIDTH + 15) / 16, (HEIGHT + 15) / 16);
    dim3 dimBlock(16, 16);

    // Launch the CUDA kernel
    convolution2D<<<dimGrid, dimBlock>>>(d_input, d_mask, d_output, WIDTH, HEIGHT, KERNEL_SIZE);

    // Copy the result back to the host
    hipMemcpy(output, d_output, WIDTH * HEIGHT * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_mask);
    hipFree(d_output);

    // Save the output image
    cv::Mat outputImage(HEIGHT, WIDTH, CV_8U);
    for (int i = 0; i < HEIGHT; i++)
    {
        for (int j = 0; j < WIDTH; j++)
        {
            outputImage.at<uchar>(i, j) = static_cast<uchar>(output[i][j]);
        }
    }
    cv::imwrite("output.jpg", outputImage);

    return 0;
}
