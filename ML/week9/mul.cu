
#include <hip/hip_runtime.h>
#include <stdio.h>

// Matrix dimensions
#define N 3
#define M 3
#define P 3

// Tile size for matrix multiplication
#define TILE_SIZE 16

// CUDA kernel to perform matrix multiplication
__global__ void matrixMultiply(int *A, int *B, int *C, int n, int m, int p)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (row < n && col < m)
    {
        for (int k = 0; k < m; k++)
        {
            int a_element = A[row * m + k];
            int b_element = B[k * p + col];
            sum += a_element * b_element;
        }

        C[row * p + col] = sum;
    }
}

int main()
{
    int a[N][M], b[M][P], c[N][P]; // Host matrices
    int *d_a, *d_b, *d_c;          // Device matrices

    // Initialize matrices a and b
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < M; j++)
        {
            a[i][j] = i * M + j;
            b[i][j] = i * M + j;
        }
    }

    // Allocate memory on the device
    hipMalloc((void **)&d_a, N * M * sizeof(int));
    hipMalloc((void **)&d_b, M * P * sizeof(int));
    hipMalloc((void **)&d_c, N * P * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_a, a, N * M * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, M * P * sizeof(int), hipMemcpyHostToDevice);

    // Calculate grid and block dimensions based on the tile size
    dim3 dimGrid((P + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);

    // Launch the CUDA kernel
    matrixMultiply<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, N, M, P);

    // Copy the result back to the host
    hipMemcpy(c, d_c, N * P * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Print the result
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < P; j++)
        {
            printf("%d\t", c[i][j]);
        }
        printf("\n");
    }

    return 0;
}
